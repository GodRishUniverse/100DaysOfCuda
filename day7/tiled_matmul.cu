
#include <hip/hip_runtime.h>
#define  TILE_WIDTH 16
__global__
void tiled_matmul_kernel(float* M, float*N, float *P, int Width){
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; // scope of shared memory is a block
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; // scope of shared memory is a block

    // one version for Mds and Nds for each block 

    // shorter name auto variables - placed in registers
    int bx = blockIdx.x; 
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // each thread is responsible for calculating one value of P element

    int Row = by * TILE_WIDTH+ ty; // vertical position - same analogy is below for the vertical position 
    int Col = bx * TILE_WIDTH +tx; // horizontal position - bx blocks of threads (bx * TILE_WIDTH) before tx (thread)

    float Pvalue {}; // automatic variable - so generated per 

    for (int p_h = 0; p_h < Width/ TILE_WIDTH; ++p_h){ // p_h is going tile by tile
        Mds[ty][tx] = M[Row*Width +p_h*TILE_WIDTH+tx];
        Nds[ty][tx] = N[(p_h*TILE_WIDTH+ty)*Width +Col];
        __syncthreads(); // used to sync the threads to do this in parallel - IN A BLOCK
    

        // This is the dot product of the row in M with the column in N
        for (int k = 0;k <TILE_WIDTH; ++k ){
            Pvalue +=Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads(); // acts as a barrier to ensure the threads are all synchorinized at this point

    }

    P[Row*Width+Col] = Pvalue;
}
