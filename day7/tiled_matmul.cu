
#include <hip/hip_runtime.h>
#define  TILE_WIDTH 16
__global__
void tiled_matmul_kernel(float* M, float*N, float *P, int Width){
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; 
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH+ ty;
    int Col = bx * TILE_WIDTH +tx;

    float Pvalue {};
    for (int p_h = 0; p_h < Width/ TILE_WIDTH; ++p_h){
        Mds[ty][tx] = M[Row*Width +p_h*TILE_WIDTH+tx];
        Nds[ty][tx] = N[(p_h*TILE_WIDTH+ty)*Width +Col];
        __syncthreads();

        for (int k = 0;k <TILE_WIDTH; ++k ){
            Pvalue +=Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads();

    }

    P[Row*Width+Col] = Pvalue;
}
