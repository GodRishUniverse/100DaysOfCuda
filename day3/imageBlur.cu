
#include <hip/hip_runtime.h>


__global__
void imageBlur(unsigned char *in, unsigned char *out, int width, int height) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    if (row < height && col < width) {
        int pixVal = 0;
        int pixels = 0;
        for (int i = -1; i <= 1; i++) { // 1 is the BLUR_SIZE
            for (int j = -1; j <= 1; j++) {
                int curRow = row + i;
                int curCol = col + j;
                if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width) {
                    pixels++;
                    pixVal += in[curRow * width + curCol];
                    
                }
            }
        }

        out[row * width + col] = (unsigned char)(pixVal / pixels);
    }
}
