
#include <hip/hip_runtime.h>
// day3/greyScale.cu

// there are 3 channels in RGB

// __global__
// void colorToGrayScale(unsigned char *P_Out, unsigned char *P_In, int width, int height) {
//     int col = blockIdx.x * blockDim.x + threadIdx.x;
//     int row = blockIdx.y * blockDim.y + threadIdx.y;
//     if (row < height && col < width) {
//         int greyOffset = row*width + col;

//         int rgbOffset = greyOffset*CHANNELS;    
//         unsigned char r = P_In[rgbOffset];
//         unsigned char g = P_In[rgbOffset + 2];
//         unsigned char b = P_In[rgbOffset + 3];
//         P_Out[greyOffset] = 0.21f *r + 0.72f*g + 0.07f*b;
//     }
// }
