
#include <hip/hip_runtime.h>
#include <iostream>

__global__
void add(float *a, float *b, float *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    c[index] = a[index] + b[index];
}

void printArray(float *array, int size) {
    for (int i = 0; i < size; i++) {
        std::cout << array[i] << " ";
    }
    std::cout << std::endl;
}

void vectorAdd(float *a, float *b, float *c, int n) {
    int size = n * sizeof(float);

    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice); // Host to Device
    hipMalloc(&d_b, size);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice); // Host to Device

    hipMalloc(&d_c, size);

    // kernel code
    add<<<ceil(n / 256.0), 256>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost); // Device to Host

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}


int main() {
    float a[] = {1.f, 2.f, 3.f, 4.f};
    float b[] = {5.f, 6.f, 7.f, 8.f};

    float c[4];

    vectorAdd(a, b, c, 4);

    printArray(c, 4);
    return 0;
}
