
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
}

void printArray(int *array, int size) {
    for (int i = 0; i < size; i++) {
        std::cout << array[i] << " ";
    }
    std::cout << std::endl;
}

void testAddition() {
    int size = 10;
    int *a, *b, *c;
    hipMallocManaged(&a, size * sizeof(int));
    hipMallocManaged(&b, size * sizeof(int));
    hipMallocManaged(&c, size * sizeof(int));

    // Initialize input arrays
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Print input arrays
    std::cout << "Array A: ";
    printArray(a, size);
    std::cout << "Array B: ";
    printArray(b, size);

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(a, b, c);

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Print result array
    std::cout << "Array C (A + B): ";
    printArray(c, size);

    // Clean up
    hipFree(a);
    hipFree(b);
    hipFree(c);
}

int main() {
    testAddition();
    return 0;
}
