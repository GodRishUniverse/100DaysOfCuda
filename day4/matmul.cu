
#include <hip/hip_runtime.h>
#include <iostream>

__global__
void kernel_matmul(float *in_a, float *in_b, float *out, int m, int n, int k) {
    // in_a is m by n
    // in_b is n by k
    // out is m by k

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        int sum {0};
        for (int i = 0; i < n; i++) {
            sum += in_a[row * n + i] * in_b[i * k + col];
        }   
        out[row * k + col] = sum;
    }
}


void matmul(float *a, float *b, float *c, int m , int n, int k) {
    int size_a = m*n * sizeof(float);
    int size_b = n*k * sizeof(float);
    int size_c = m*k * sizeof(float);

    float *d_a, *d_b, *d_c;

    hipError_t err = hipMalloc((void **)&d_a, size_a);
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err), __FILE__,__LINE__);
        exit(EXIT_FAILURE);
    }
    hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice); // Host to Device
    err = hipMalloc((void**)&d_b, size_b);
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err), __FILE__,__LINE__);
        exit(EXIT_FAILURE);
    }
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice); // Host to Device

    err = hipMalloc((void **)&d_c, size_c);
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err), __FILE__,__LINE__);
        exit(EXIT_FAILURE);
    }

    // kernel code
    dim3 dimGrid(32, 1,1);
    dim3 dimBlock(32, 1,1);
    kernel_matmul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);

    hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost); // Device to Host

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}



int main(){
    // we will test here

    float a[] = {1.f, 2.f, 3.f, 4.f};
    float b[] = {5.f, 6.f, 7.f, 8.f};

    float c[1];

    matmul(a, b, c, 1, 4, 1);
    std::cout << c[0] << std::endl;

    return 0;

    
}
