
#include <hip/hip_runtime.h>
#define  TILE_WIDTH 16
__global__
void arb_tiled_matmul_kernel(float* M, float*N, float *P, int Width){
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; 

  
    int bx = blockIdx.x; 
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;


    int Row = by * TILE_WIDTH+ ty;  
    int Col = bx * TILE_WIDTH +tx; 
    float Pvalue {};

    for (int p_h = 0; p_h < Width/ TILE_WIDTH; ++p_h){
        if (Row < Width && (p_h*TILE_WIDTH+tx)< Width){
            Mds[ty][tx] = M[Row*Width +p_h*TILE_WIDTH+tx];
        }else{
            Mds[ty][tx] = 0.0f; // safest value if used in calculation
        }

        if ((p_h*TILE_WIDTH+ty)<Width && Col<Width){
            Nds[ty][tx] = N[(p_h*TILE_WIDTH+ty)*Width +Col];
        } else{
            Nds[ty][tx] = 0.0f; // will not cause harm to the data
        }
        
        __syncthreads();
    
        for (int k = 0;k <TILE_WIDTH; ++k ){
            Pvalue +=Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads();

    }
    if ((Row <Width) && (Col <Width)){
        P[Row*Width+Col] = Pvalue;
    }
    
}
