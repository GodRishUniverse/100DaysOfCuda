
#include <hip/hip_runtime.h>
#include <iostream>

#define  TILE_WIDTH 16

#define ll long long

// Matrices be j*c and c*l
__global__
void gen_tiled_matmul_kernel(float* M, float*N, float *P, int j , int c , int l){
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; 

  
    int bx = blockIdx.x; 
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;


    int Row = by * TILE_WIDTH+ ty;  
    int Col = bx * TILE_WIDTH +tx; 
    float Pvalue {};

    // ceil ensures that the upper end of the tile is chosen
    for (int p_h = 0; p_h < static_cast<int>(ceil(c / TILE_WIDTH)); ++p_h){
        if (Row < j && (p_h*TILE_WIDTH+tx)< c){
            Mds[ty][tx] = M[Row*c +p_h*TILE_WIDTH+tx];
        }else{
            Mds[ty][tx] = 0.0f; // safest value if used in calculation
        }

        if ((p_h*TILE_WIDTH+ty)<c && Col<l){
            Nds[ty][tx] = N[(p_h*TILE_WIDTH+ty)*l +Col];
        } else{
            Nds[ty][tx] = 0.0f; // will not cause harm to the data
        }
        
        __syncthreads();
    
        for (int k = 0;k <TILE_WIDTH; ++k ){
            Pvalue +=Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads();

    }
    if ((Row <j) && (Col <l )){
        P[Row*l+Col] = Pvalue;
    }
    
}


void execute_mul(float * M, float *N, float *P, int j, int c, int l ){
    ll size_m, size_n, size_p;
    size_m = j*c*sizeof(float);
    size_n = c*l*sizeof(float);
    size_p = j*l*sizeof(float);


    float* d_M;
    float* d_N;
    float* d_P;

    hipError_t error = hipMalloc((void**) &d_M, size_m);
    if (error !=hipSuccess){
        printf("Error: %s\n", hipGetErrorString(error), __FILE__,__LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_M, M, size_m, hipMemcpyHostToDevice);


    error = hipMalloc((void**) &d_N, size_n);
    if (error !=hipSuccess){
        printf("Error: %s\n", hipGetErrorString(error), __FILE__,__LINE__);
        exit(EXIT_FAILURE);
    }

    hipMemcpy(d_N, N, size_n, hipMemcpyHostToDevice);


    error = hipMalloc((void**) &d_P, size_p);
    if (error !=hipSuccess){
        printf("Error: %s\n", hipGetErrorString(error), __FILE__,__LINE__);
        exit(EXIT_FAILURE);
    }

    // TODO: need to see if this is correct or not
    gen_tiled_matmul_kernel<<<4, 256>>> (d_M, d_N, d_P, j,c ,l);

    hipMemcpy(P, d_P, size_p, hipMemcpyDeviceToHost);

    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

}

int main(){
    
}
